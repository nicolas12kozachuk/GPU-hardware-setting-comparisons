#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <stdint.h>
#include <cstdint>

#define N  256

// function to check for CUDA errors
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

// kernel to perform 2D matrix multiplication
__global__ void matrixMulGPU( int * a, int * b, int * c)
{

  int indexWithinTheGridX = threadIdx.x + blockIdx.x * blockDim.x;
  int gridStrideX = gridDim.x * blockDim.x;
  
  int indexWithinTheGridY = threadIdx.y + blockIdx.y * blockDim.y;
  int gridStrideY = gridDim.y * blockDim.y;

  for (int i = indexWithinTheGridX; i < N; i += gridStrideX)
  {
    for (int j = indexWithinTheGridY; j < N; j += gridStrideY)
      {
        int val = 0;
        for ( int k = 0; k < N; ++k ){
            val += a[(i * N + k)] * b[(k * N + j)];
        }
        c[i * N + j] =  val;
      }
  }
}

/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  clock_t start_time = clock();

  int *a, *b, *c_cpu, *c_gpu, *data_gpu, *data_cpu; // Allocate a solution matrix for both the CPU and the GPU operations

  const uint64_t size_r = N*N; 
  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  hipHostMalloc(&data_cpu, size);
  hipMalloc    (&data_gpu, size);

  // Initialize memory; create 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
      data_cpu[row*N + col] = 0;
    }

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  const uint64_t num_streams = 2;

  hipStream_t streams[num_streams];
  for (uint64_t stream = 0; stream < num_streams; stream++)
      hipStreamCreate(&streams[stream]);

  dim3 threads_per_block(16,16,1); 
  dim3 number_of_blocks(16,16,1);

  int x = size_r;
  int y = num_streams;
  const uint64_t chunk_size = x/y + (x % y != 0);

  for (uint64_t stream = 0; stream < num_streams; stream++) {

    const uint64_t lower = chunk_size*stream;
    // For tail stream `lower+chunk_size` could be out of range, so here we guard against that.
    const uint64_t upper = min(lower+chunk_size, size_r);
    // Since the tail stream width may not be `chunk_size`,
    // we need to calculate a separate `width` value.
    const uint64_t width = upper-lower;

    // copy memory from host to device
    hipMemcpyAsync(data_gpu+lower, data_cpu+lower, 
           sizeof(int)*width, hipMemcpyHostToDevice, 
           streams[stream]);

    matrixMulGPU <<< number_of_blocks, threads_per_block, 0, streams[stream]>>> ( a, b, data_gpu);

    checkCuda( hipGetLastError() );
    checkCuda( hipDeviceSynchronize() );

    //copy memory from device to host
    hipMemcpyAsync(data_cpu+lower, data_gpu+lower, 
           sizeof(int)*width, hipMemcpyDeviceToHost, 
           streams[stream]);
  }


  // Destroy streams.
  for (uint64_t stream = 0; stream < num_streams; stream++)
      hipStreamDestroy(streams[stream]);

  clock_t end_time = clock();

  float execution_time = ((float)(end_time - start_time)) / CLOCKS_PER_SEC;
  printf("Execution Time: %f seconds\n", execution_time);

  
  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != data_cpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");
    
  // Free all our allocated memory
  hipFree( c_cpu );
  hipFree(a); hipFree(b);
  hipFree( c_gpu );
  hipFree(data_cpu);
  hipFree(data_gpu);
}
